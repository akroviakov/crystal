#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

/**
 * Globals, constants and typedefs
 */
bool                    g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

enum QueryVariant {
    Vector = 0,
    Vector_opt = 1,
    Compiled = 2
};

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void DeviceSelectIfCompiled(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice,
    int lo_num_entries, unsigned long long* revenue, int batchId=-1) {

  long long sum = 0;
  int blockIndex = (batchId == -1) ? blockIdx.x : batchId + blockIdx.x;
  int tile_offset = blockIndex * TILE_SIZE;
  int num_tiles = (lo_num_entries + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;
  if (blockIndex == num_tiles - 1) {
    num_tile_items = lo_num_entries - tile_offset;
  }

  for(int i = 0; i < ITEMS_PER_THREAD; i++){
    if(threadIdx.x + i * BLOCK_THREADS < num_tile_items){
      int offset = tile_offset + threadIdx.x + BLOCK_THREADS * i;
      if(offset < lo_num_entries){
        if(lo_orderdate[offset] >= 19940101 && lo_orderdate[offset] <= 19940131 && 
            lo_quantity[offset] >= 26 && lo_quantity[offset] <= 35 && 
            lo_discount[offset] >= 4 && lo_discount[offset] <= 6){
          sum += lo_discount[offset] * lo_extendedprice[offset];
        }
      }
    }
  }
  __syncthreads();
  static __shared__ long long buffer[32];
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(revenue, aggregate);
  }
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD, QueryVariant QImpl>
__global__ void DeviceSelectIf(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice,
    int lo_num_entries, unsigned long long* revenue) {
  // Load a segment of consecutive items that are blocked across threads
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];

  long long sum = 0;

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (lo_num_entries + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = lo_num_entries - tile_offset;
  }
  if constexpr (QImpl == QueryVariant::Vector){
    BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset, items, num_tile_items);
    BlockPredGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 19940101, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 19940131, selection_flags, num_tile_items);

    BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_quantity + tile_offset, items, num_tile_items);
    BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 26, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 35, selection_flags, num_tile_items);

    BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_discount + tile_offset, items, num_tile_items);
    BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 4, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 6, selection_flags, num_tile_items);

    BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_extendedprice + tile_offset, items2, num_tile_items);
  } else {
    BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset,
                                                    items, num_tile_items);
    BlockPredGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 19940101, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 19940131, selection_flags, num_tile_items);

    BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        lo_quantity + tile_offset, items, num_tile_items, selection_flags);
    BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 26, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 35, selection_flags, num_tile_items);

    BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        lo_discount + tile_offset, items, num_tile_items, selection_flags);
    BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 4, selection_flags, num_tile_items);
    BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        items, 6, selection_flags, num_tile_items);

    BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
        lo_extendedprice + tile_offset, items2, num_tile_items, selection_flags);
  }
  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
  {
    if (threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items)
      if (selection_flags[ITEM])
        sum += items[ITEM] * items2[ITEM];
  }

  __syncthreads();

  static __shared__ long long buffer[32];
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(revenue, aggregate);
  }
}


template<QueryVariant QImpl>
float runQuery(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice, 
    int lo_num_entries, hipcub::CachingDeviceAllocator&  g_allocator) {
  SETUP_TIMING();

  float time_query;
  chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();

  hipEventRecord(start, 0);

  unsigned long long* d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_sum, sizeof(long long)));

  hipMemset(d_sum, 0, sizeof(long long));

  // Run
  if constexpr(QImpl == QueryVariant::Vector || QImpl == QueryVariant::Vector_opt){
  int tile_items = 128*4;
  DeviceSelectIf<128,4,QImpl><<<(lo_num_entries + tile_items - 1)/tile_items, 128>>>(lo_orderdate, 
          lo_discount, lo_quantity, lo_extendedprice, lo_num_entries, d_sum);
  } else {
    constexpr int batchSize{20000};
    constexpr int numBatches{(LO_LEN + batchSize - 1) / batchSize};
    constexpr int numThreads{1024};
    constexpr int elemPerThread{batchSize / numThreads + 1};
    if constexpr(QImpl == QueryVariant::Compiled){
      constexpr int numBlocks{numBatches};
      DeviceSelectIfCompiled<numThreads,elemPerThread><<<numBlocks, numThreads>>>(lo_orderdate, 
          lo_discount, lo_quantity, lo_extendedprice, lo_num_entries, d_sum);
    } else {

    }
  }  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_query, start,stop);

  unsigned long long revenue;
  CubDebugExit(hipMemcpy(&revenue, d_sum, sizeof(long long), hipMemcpyDeviceToHost));

  finish = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = finish - st;

  cout << "Revenue: " << revenue << endl;
  cout << "Time Taken Total: " << diff.count() * 1000 << endl;

  CLEANUP(d_sum);

  return time_query;
}

/**
 * Main
 */
int main(int argc, char** argv)
{
  int num_trials          = 10;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("t", num_trials);
  string dataSetPath;
  args.GetCmdLineArgument("dataSetPath", dataSetPath);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
      printf("%s "
          "[--t=<num trials>] "
          "[--v] "
          "\n", argv[0]);
      exit(0);
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  int *h_lo_orderdate = loadColumn<int>(dataSetPath,"lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>(dataSetPath,"lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>(dataSetPath,"lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>(dataSetPath,"lo_extendedprice", LO_LEN);
  int *h_d_datekey = loadColumn<int>(dataSetPath,"d_datekey", D_LEN);
  int *h_d_year = loadColumn<int>(dataSetPath,"d_year", D_LEN);

  cout << "** LOADED DATA **" << endl;

  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice = loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);
  int *d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
  int *d_d_year = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

  cout << "** LOADED DATA TO GPU **" << endl;

  cout << "** VECTOR TEST **" << endl;
  for (int t = 0; t < num_trials; t++) {
    float time_query;
    time_query = runQuery<QueryVariant::Vector>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, LO_LEN, g_allocator);
    cout<< "{"
        << "\"type\":vec" 
        << ",\"query\":12" 
        << ",\"time_query\":" << time_query
        << "}" << endl;
  }
  cout << "** VECTOR-OPT TEST **" << endl;
  for (int t = 0; t < num_trials; t++) {
    float time_query;
    time_query = runQuery<QueryVariant::Vector_opt>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, LO_LEN, g_allocator);
    cout<< "{"
        << "\"type\":vecOpt" 
        << ",\"query\":12" 
        << ",\"time_query\":" << time_query
        << "}" << endl;
  }

  cout << "** COMPILED TEST **" << endl;
  for (int t = 0; t < num_trials; t++) {
    float time_query;
    time_query = runQuery<QueryVariant::Compiled>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, LO_LEN, g_allocator);
    cout<< "{"
        << "\"type\":comp" 
        << ",\"query\":12" 
        << ",\"time_query\":" << time_query
        << "}" << endl;
  }
  return 0;
}

